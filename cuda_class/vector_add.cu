
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>

__global__ void add(int n, float *x, float *y) {
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
}

void FillWithData(int n, float* x, float* y) {
  for (int i = 0; i < n; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
} 

int main(void) {
  int N = 1<<20;
  float *x, *y;
  float *d_x, *d_y;
  int size = N * sizeof(float);

  x = (float*) malloc(size);
  y = (float*) malloc(size);
  FillWithData(N, x, y);

  hipMalloc(&d_x, size);
  hipMalloc(&d_y, size);
  hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, size, hipMemcpyHostToDevice); 

  add<<<2, 256>>>(N, d_x, d_y);

  hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost);
  hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost); 

  int i = 0;
  int sample_rate = N / 10;
  for (i = 0; i < N; i=i+sample_rate) {
    printf("Value y[%d] = %f\n" , i, y[i]);
  } 

  // Free memory
  free(x); free(y);
  hipFree(d_x); hipFree(d_y);
  
  return 0;
}
